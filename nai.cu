#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_vector_types.h"
#include <stdio.h>
#include <iostream>

#define CacheCount 3

__global__ void BenchMarkDRAMKernel(float4* In) {
    int ThreadID = blockDim.x *blockIdx.x + threadIdx.x ;

    float4 Temp = make_float4(1);

    Temp += In[ThreadID];

    if (length(Temp) == -12354) {
        In[0] = Temp;
    }
}

__global__ void BenchMarkCacheKernel(float4* In, int Zero) {
    int ThreadID = blockDim.x *blockIdx.x + threadIdx.x;

    float4 Temp = make_float4(1);

    #pragma unroll
    for (int i = 0; i < CacheCount; i++) {
        Temp += In[ThreadID + i*Zero];
    }

    if (length(Temp) == -12354) {
        In[0] = Temp;
    }
}

int main() {
    static const int PointerCount = 5000;

    int Float4Count = 4 * 1024 * 1024;
    int ChunkSize = Float4Count*sizeof(float4);
    float4* Pointers[PointerCount];
    int UsedPointers = 0;
    printf("Nai's Benchmark \n");
    printf("Allocating Memory . . . Chunk Size = %i Byte \n", ChunkSize);
    system("pause");

    while (true) {
        int Error = hipMalloc(&Pointers[UsedPointers], ChunkSize);

        if (Error == hipErrorOutOfMemory)
        break;

        printf("Allocated at %llx\n", (long long unsigned)Pointers[UsedPointers]);

        hipMemset(Pointers[UsedPointers], 0, ChunkSize);
        UsedPointers++;
    }

    printf("Allocated %i Chunks \n", UsedPointers);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int BlockSize = 128;
    int BlockCount = Float4Count / BlockSize;

    int BenchmarkCount = 10;

    printf("Benchmarking DRAM \n");

    for (int i = 0; i < UsedPointers; i++) {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++)
            BenchMarkDRAMKernel <<<BlockCount, BlockSize >>>(Pointers[i]);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        float Bandwidth = ((float)(BenchmarkCount * ChunkSize)) / milliseconds/ 1000.f/1000.f;
        printf("DRAM-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    printf("Copying between low and low chunks\n");

    for (int i = 0; i < 10; i++) {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++) {
            hipMemcpy(Pointers[i], Pointers[i + 10], ChunkSize, hipMemcpyDeviceToDevice);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        float Bandwidth = ((float)(BenchmarkCount * ChunkSize)) / ms/ 1000.f/1000.f;
        printf("DRAM-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    printf("Copying between high and low chunks\n");

    for (int i = 0; i < 10; i++) {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++) {
            hipMemcpy(Pointers[i], Pointers[UsedPointers - i - 1], ChunkSize, hipMemcpyDeviceToDevice);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        float Bandwidth = ((float)(BenchmarkCount * ChunkSize)) / ms/ 1000.f/1000.f;
        printf("DRAM-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    printf("Copying between high and high chunks\n");

    for (int i = 0; i < 4; i++) {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++) {
            hipMemcpy(Pointers[UsedPointers - i - 5], Pointers[UsedPointers - i - 1], ChunkSize, hipMemcpyDeviceToDevice);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float ms = 0;
        hipEventElapsedTime(&ms, start, stop);
        float Bandwidth = ((float)(BenchmarkCount * ChunkSize)) / ms/ 1000.f/1000.f;
        printf("DRAM-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    printf("Freeing first chunks\n");

    // Free the first half of memory
    for (int i = 0; i < UsedPointers / 2; i++) {
        int error = hipFree(Pointers[UsedPointers]);
        if (error != hipSuccess) {
            printf("Error freeing memory\n");
            exit(-1);
        }
    }

    printf("Re-benchmarking upper half of memory\n");

    for (int i = UsedPointers / 2; i < UsedPointers; i++) {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++)
        BenchMarkDRAMKernel <<<BlockCount, BlockSize >>>(Pointers[i]);

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        float Bandwidth = ((float)(BenchmarkCount * ChunkSize)) / milliseconds/ 1000.f/1000.f;
        printf("DRAM-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    printf("Exiting early.");
    exit(0);

    printf("Benchmarking L2-Cache \n");

    for (int i = 0; i < UsedPointers; i++)
    {
        hipEventRecord(start);
        for (int j = 0; j < BenchmarkCount; j++) {
            BenchMarkCacheKernel << <BlockCount, BlockSize >> >(Pointers[i], 0);
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        float Bandwidth = (((float)CacheCount* (float)BenchmarkCount * (float)ChunkSize)) / milliseconds / 1000.f / 1000.f;
        printf("L2-Cache-Bandwidth of %i. Chunk: %f GByte/s \n", i, Bandwidth);
    }

    hipDeviceSynchronize();
    hipDeviceReset();
    return 0;
}
